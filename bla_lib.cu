#include "hip/hip_runtime.h"
/* CUDA tutorial: Basic Linear Algebra (BLA) Library

!Copyright (C) 2018-2018 Dmitry I. Lyakh (Liakh)
!Copyright (C) 2018-2018 Oak Ridge National Laboratory (UT-Battelle)

!This file is part of CUDA BLA tutorial.

!CUDA BLA is free software: you can redistribute it and/or modify
!it under the terms of the GNU Lesser General Public License as published
!by the Free Software Foundation, either version 3 of the License, or
!(at your option) any later version.

!CUDA BLA is distributed in the hope that it will be useful,
!but WITHOUT ANY WARRANTY; without even the implied warranty of
!MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
!GNU Lesser General Public License for more details.

!You should have received a copy of the GNU Lesser General Public License
!along with CUDA BLA. If not, see <http://www.gnu.org/licenses/>. */

#include <stdio.h>
#include <iostream>
#include <assert.h>

#include "bla_lib.hpp"

namespace bla{

//Number of present GPU devices:
static int gpuAmount = 0;

//CUDA device properties:
hipDeviceProp_t * gpuProperty;

//cuBLAS handles (one per device):
hipblasHandle_t * cublasHandle;

//CUDA kernel prototypes:
template <typename T>
__global__ void gpu_array_norm(size_t arr_size, const T * __restrict__ arr, volatile T * norm);
__device__ static unsigned int norm_wr_lock = 0; //reduction lock (per GPU)

template <typename T>
__global__ void gpu_array_add(size_t arr_size, T * __restrict__ arr0, const T * __restrict__ arr1);

template <typename T>
__global__ void gpu_gemm_nn(int m, int n, int k, T * __restrict__ dest, const T * __restrict__ left, const T * __restrict__ right);
static const int TILE_EXT_X = 16;
static const int TILE_EXT_Y = 16;


//DEFINITIONS:
void init()
{
 gpuAmount=0;
 hipError_t cuerr = hipGetDeviceCount(&gpuAmount); assert(cuerr == hipSuccess);
 std::cout << "Found " << gpuAmount << " NVIDIA GPU" << std::endl;
 if(gpuAmount > 0){
  hipblasStatus_t cuberr;
  gpuProperty = new hipDeviceProp_t[gpuAmount];
  cublasHandle = new hipblasHandle_t[gpuAmount];
  //Init each GPU:
  for(int i = gpuAmount-1; i >= 0; --i){
   cuerr = hipSetDevice(i); assert(cuerr == hipSuccess);
   cuerr = hipGetDeviceProperties(&(gpuProperty[i]),i); assert(cuerr == hipSuccess);
   cuberr = hipblasCreate(&(cublasHandle[i])); assert(cuberr == HIPBLAS_STATUS_SUCCESS);
   cuberr = hipblasSetPointerMode(cublasHandle[i],HIPBLAS_POINTER_MODE_DEVICE); assert(cuberr == HIPBLAS_STATUS_SUCCESS);
   std::cout << "Initialized GPU " << i << std::endl;
  }
  //Enable P2P access between GPU:
  for(int i = gpuAmount-1; i >= 0; --i){
   if(gpuProperty[i].unifiedAddressing != 0){
    cuerr = hipSetDevice(i); assert(cuerr == hipSuccess);
    for(int j = gpuAmount-1; j >= 0; --j){
     if(j != i){
      if(gpuProperty[j].unifiedAddressing != 0){
       cuerr = hipDeviceEnablePeerAccess(j,0); assert(cuerr == hipSuccess);
       std::cout << "GPU " << i << " can access peer GPU " << j << std::endl;
      }
     }
    }
   }
  }
 }
 std::cout << "BLA library initialized successfully" << std::endl;
 return;
}


void shutdown()
{
 if(gpuAmount > 0){
  hipError_t cuerr;
  hipblasStatus_t cuberr;
  for(int i = 0; i < gpuAmount; ++i){
   cuberr = hipblasDestroy(cublasHandle[i]); assert(cuberr == HIPBLAS_STATUS_SUCCESS);
   cuerr = hipDeviceReset(); assert(cuerr == hipSuccess);
   std::cout << "Destroyed primary context for GPU " << i << std::endl;
  }
  delete [] cublasHandle;
  delete [] gpuProperty;
 }
 gpuAmount=0;
 std::cout << "BLA library shut down successfully" << std::endl;
 return;
}


template <typename T>
__global__ void gpu_array_norm(size_t arr_size, const T * __restrict__ arr, volatile T * norm)
{
 extern __shared__ T thread_norm[]; //blockDim.x

 size_t n = gridDim.x*blockDim.x;
 T tnorm = static_cast<T>(0.0);
 for(size_t i = blockIdx.x*blockDim.x + threadIdx.x; i < arr_size; i += n) tnorm += arr[i] * arr[i];
 thread_norm[threadIdx.x] = tnorm;
 __syncthreads();

 unsigned int s = blockDim.x;
 while(s > 1){
  unsigned int j = (s+1U)>>1; //=(s+1)/2
  if(threadIdx.x + j < s) thread_norm[threadIdx.x] += thread_norm[threadIdx.x+j];
  __syncthreads();
  s=j;
 }

 if(threadIdx.x == 0){
  unsigned int j = 1;
  while(j){j = atomicMax(&norm_wr_lock,1);} //lock
  *norm += thread_norm[0]; //accumulate
  __threadfence();
  j=atomicExch(&norm_wr_lock,0); //unlock
 }
 __syncthreads();
 return;
}


template <typename T>
__global__ void gpu_array_add(size_t arr_size, T * __restrict__ arr0, const T * __restrict__ arr1)
{
 size_t n = gridDim.x * blockDim.x;
 for(size_t i = blockIdx.x*blockDim.x + threadIdx.x; i < arr_size; i += n) arr0[i] += arr1[i];
 return;
}


template <typename T>
__global__ void gpu_gemm_nn(int m, int n, int k, T * __restrict__ dest, const T * __restrict__ left, const T * __restrict__ right)
{
 T __shared__ lbuf[TILE_EXT_X][TILE_EXT_Y],rbuf[TILE_EXT_X][TILE_EXT_Y];

 return;
}

} //namespace bla
